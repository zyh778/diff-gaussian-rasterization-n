#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2023, Inria
 * GRAPHDECO research group, https://team.inria.fr/graphdeco
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use 
 * under the terms of the LICENSE.md file.
 *
 * For inquiries contact  george.drettakis@inria.fr
 */

#include "forward.h"
#include "auxiliary.h"
#include "hip/hip_vector_types.h"
#include "math.h"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
namespace cg = cooperative_groups;

// 将球谐函数系数转换为RGB颜色的前向传播函数
// idx: 高斯体索引
// deg: 球谐函数的度数
// max_coeffs: 最大系数数量
// means: 高斯体中心点位置
// campos: 相机位置
// shs: 球谐函数系数
// clamped: 记录是否被截断到0的标志
__device__ glm::vec3 computeColorFromSH(int idx, int deg, int max_coeffs, const glm::vec3* means, glm::vec3 campos, const float* shs, bool* clamped)
{
	// 基于Zhang等人2022年论文"Differentiable Point-Based Radiance Fields for Efficient View Synthesis"的实现
	glm::vec3 pos = means[idx];
	// 计算从相机到高斯体的方向向量并归一化
	glm::vec3 dir = pos - campos;
	dir = dir / glm::length(dir);

	// 获取当前高斯体的球谐系数
	glm::vec3* sh = ((glm::vec3*)shs) + idx * max_coeffs;
	// 计算0阶球谐函数的贡献
	glm::vec3 result = SH_C0 * sh[0];

	// 计算更高阶球谐函数的贡献
	if (deg > 0)
	{
		float x = dir.x;
		float y = dir.y;
		float z = dir.z;
		// 1阶球谐函数
		result = result - SH_C1 * y * sh[1] + SH_C1 * z * sh[2] - SH_C1 * x * sh[3];

		if (deg > 1)
		{
			float xx = x * x, yy = y * y, zz = z * z;
			float xy = x * y, yz = y * z, xz = x * z;
			// 2阶球谐函数
			result = result +
				SH_C2[0] * xy * sh[4] +
				SH_C2[1] * yz * sh[5] +
				SH_C2[2] * (2.0f * zz - xx - yy) * sh[6] +
				SH_C2[3] * xz * sh[7] +
				SH_C2[4] * (xx - yy) * sh[8];

			if (deg > 2)
			{
				// 3阶球谐函数
				result = result +
					SH_C3[0] * y * (3.0f * xx - yy) * sh[9] +
					SH_C3[1] * xy * z * sh[10] +
					SH_C3[2] * y * (4.0f * zz - xx - yy) * sh[11] +
					SH_C3[3] * z * (2.0f * zz - 3.0f * xx - 3.0f * yy) * sh[12] +
					SH_C3[4] * x * (4.0f * zz - xx - yy) * sh[13] +
					SH_C3[5] * z * (xx - yy) * sh[14] +
					SH_C3[6] * x * (xx - 3.0f * yy) * sh[15];
			}
		}
	}
	result += 0.5f;

	// 将RGB颜色限制在正值范围内,并记录是否被截断
	clamped[3 * idx + 0] = (result.x < 0);
	clamped[3 * idx + 1] = (result.y < 0);
	clamped[3 * idx + 2] = (result.z < 0);
	return glm::max(result, 0.0f);
}

// 计算2D协方差矩阵的前向传播函数
__device__ float3 computeCov2D(const float3& mean, float focal_x, float focal_y, float tan_fovx, float tan_fovy, const float* cov3D, const float* viewmatrix)
{
	// 基于Zwicker等人2002年论文"EWA Splatting"中的公式29和31
	// 同时考虑了视口的缩放比例
	// 将点变换到相机空间
	float3 t = transformPoint4x3(mean, viewmatrix);

	// 限制投影点在视锥体内
	const float limx = 1.3f * tan_fovx;
	const float limy = 1.3f * tan_fovy;
	const float txtz = t.x / t.z;
	const float tytz = t.y / t.z;
	t.x = min(limx, max(-limx, txtz)) * t.z;
	t.y = min(limy, max(-limy, tytz)) * t.z;

	// 计算雅可比矩阵J
	glm::mat3 J = glm::mat3(
		focal_x / t.z, 0.0f, -(focal_x * t.x) / (t.z * t.z),
		0.0f, focal_y / t.z, -(focal_y * t.y) / (t.z * t.z),
		0, 0, 0);

	// 视图矩阵W
	glm::mat3 W = glm::mat3(
		viewmatrix[0], viewmatrix[4], viewmatrix[8],
		viewmatrix[1], viewmatrix[5], viewmatrix[9],
		viewmatrix[2], viewmatrix[6], viewmatrix[10]);

	// 计算变换矩阵T
	glm::mat3 T = W * J;

	// 构建3D协方差矩阵
	glm::mat3 Vrk = glm::mat3(
		cov3D[0], cov3D[1], cov3D[2],
		cov3D[1], cov3D[3], cov3D[4],
		cov3D[2], cov3D[4], cov3D[5]);

	// 计算2D协方差矩阵
	glm::mat3 cov = glm::transpose(T) * glm::transpose(Vrk) * T;

	// 应用低通滤波:确保每个高斯体至少有1个像素的宽度/高度
	cov[0][0] += 0.3f;
	cov[1][1] += 0.3f;
	return { float(cov[0][0]), float(cov[0][1]), float(cov[1][1]) };
}

// 计算3D协方差矩阵的前向传播函数
__device__ void computeCov3D(const glm::vec3 scale, float mod, const glm::vec4 rot, float* cov3D)
{
	// 创建缩放矩阵
	glm::mat3 S = glm::mat3(1.0f);
	S[0][0] = mod * scale.x;
	S[1][1] = mod * scale.y;
	S[2][2] = mod * scale.z;

	// 获取四元数参数
	glm::vec4 q = rot;
	float r = q.x;
	float x = q.y;
	float y = q.z;
	float z = q.w;

	// 从四元数计算旋转矩阵
	glm::mat3 R = glm::mat3(
		1.f - 2.f * (y * y + z * z), 2.f * (x * y - r * z), 2.f * (x * z + r * y),
		2.f * (x * y + r * z), 1.f - 2.f * (x * x + z * z), 2.f * (y * z - r * x),
		2.f * (x * z - r * y), 2.f * (y * z + r * x), 1.f - 2.f * (x * x + y * y)
	);

	// 计算变换矩阵M
	glm::mat3 M = S * R;

	// 计算3D世界空间协方差矩阵Sigma
	glm::mat3 Sigma = glm::transpose(M) * M;

	// 由于协方差矩阵是对称的,只存储上三角部分
	cov3D[0] = Sigma[0][0];
	cov3D[1] = Sigma[0][1];
	cov3D[2] = Sigma[0][2];
	cov3D[3] = Sigma[1][1];
	cov3D[4] = Sigma[1][2];
	cov3D[5] = Sigma[2][2];
}

// 光栅化前的预处理CUDA核函数
template<int C>
__global__ void preprocessCUDA(int P, int D, int M,
	const float* orig_points,
	const glm::vec3* scales,
	const float scale_modifier,
	const glm::vec4* rotations,
	const float* opacities,
	const float* shs,
	bool* clamped,
	const float* cov3D_precomp,
	const float* colors_precomp,
	const float* viewmatrix,
	const float* projmatrix,
	const glm::vec3* cam_pos,
	const int W, int H,
	const float tan_fovx, float tan_fovy,
	const float focal_x, float focal_y,
	int* radii,
	float2* points_xy_image,
	float* depths,
	float* cov3Ds,
	float* rgb,
	float4* conic_opacity,
	const dim3 grid,
	uint32_t* tiles_touched,
	bool prefiltered)
{
	// 获取当前线程的索引
	auto idx = cg::this_grid().thread_rank();
	if (idx >= P)
		return;

	// 初始化半径和触及的瓦片数为0
	radii[idx] = 0;
	tiles_touched[idx] = 0;

	// 执行近平面剔除
	float3 p_view;
	if (!in_frustum(idx, orig_points, viewmatrix, projmatrix, prefiltered, p_view))
		return;

	// 将点投影到屏幕空间
	float3 p_orig = { orig_points[3 * idx], orig_points[3 * idx + 1], orig_points[3 * idx + 2] };
	float4 p_hom = transformPoint4x4(p_orig, projmatrix);
	float p_w = 1.0f / (p_hom.w + 0.0000001f);
	float3 p_proj = { p_hom.x * p_w, p_hom.y * p_w, p_hom.z * p_w };

	// 获取或计算3D协方差矩阵
	const float* cov3D;
	if (cov3D_precomp != nullptr)
	{
		cov3D = cov3D_precomp + idx * 6;
	}
	else
	{
		computeCov3D(scales[idx], scale_modifier, rotations[idx], cov3Ds + idx * 6);
		cov3D = cov3Ds + idx * 6;
	}

	// 计算2D屏幕空间协方差矩阵
	float3 cov = computeCov2D(p_orig, focal_x, focal_y, tan_fovx, tan_fovy, cov3D, viewmatrix);

	// 计算协方差矩阵的逆(EWA算法)
	float det = (cov.x * cov.z - cov.y * cov.y);
	if (det == 0.0f)
		return;
	float det_inv = 1.f / det;
	float3 conic = { cov.z * det_inv, -cov.y * det_inv, cov.x * det_inv };

	// 计算屏幕空间范围并确定边界矩形
	float mid = 0.5f * (cov.x + cov.z);
	float lambda1 = mid + sqrt(max(0.1f, mid * mid - det));
	float lambda2 = mid - sqrt(max(0.1f, mid * mid - det));
	float my_radius = ceil(3.f * sqrt(max(lambda1, lambda2)));
	float2 point_image = { ndc2Pix(p_proj.x, W), ndc2Pix(p_proj.y, H) };
	uint2 rect_min, rect_max;
	getRect(point_image, my_radius, rect_min, rect_max, grid);
	if ((rect_max.x - rect_min.x) * (rect_max.y - rect_min.y) == 0)
		return;

	// 计算或使用预计算的颜色
	if (colors_precomp == nullptr)
	{
		glm::vec3 result = computeColorFromSH(idx, D, M, (glm::vec3*)orig_points, *cam_pos, shs, clamped);
		rgb[idx * C + 0] = result.x;
		rgb[idx * C + 1] = result.y;
		rgb[idx * C + 2] = result.z;
	}

	// 存储辅助数据供后续使用
	depths[idx] = p_view.z;
	radii[idx] = my_radius;
	points_xy_image[idx] = point_image;
	// 将2D协方差矩阵的逆和不透明度打包到一个float4中
	conic_opacity[idx] = { conic.x, conic.y, conic.z, opacities[idx] };
	tiles_touched[idx] = (rect_max.y - rect_min.y) * (rect_max.x - rect_min.x);
}

// 主光栅化CUDA核函数
template <uint32_t CHANNELS>
__global__ void __launch_bounds__(BLOCK_X * BLOCK_Y)
renderCUDA(
	const uint2* __restrict__ ranges,
	const uint32_t* __restrict__ point_list,
	int W, int H,
	const float2* __restrict__ points_xy_image,
	const float* __restrict__ features,
	const float4* __restrict__ conic_opacity,
	float* __restrict__ final_T,
	uint32_t* __restrict__ n_contrib,
	const float* __restrict__ bg_color,
	float* __restrict__ out_color,
	const float* __restrict__ depth,
	float* __restrict__ out_depth, 
	float* __restrict__ out_opacity,
	int * __restrict__ is_used)
{
	// 获取当前线程块和像素信息
	auto block = cg::this_thread_block();
	uint32_t horizontal_blocks = (W + BLOCK_X - 1) / BLOCK_X;
	uint2 pix_min = { block.group_index().x * BLOCK_X, block.group_index().y * BLOCK_Y };
	uint2 pix_max = { min(pix_min.x + BLOCK_X, W), min(pix_min.y + BLOCK_Y , H) };
	uint2 pix = { pix_min.x + block.thread_index().x, pix_min.y + block.thread_index().y };
	uint32_t pix_id = W * pix.y + pix.x;
	float2 pixf = { (float)pix.x, (float)pix.y };

	// 检查线程是否对应有效像素
	bool inside = pix.x < W&& pix.y < H;
	bool done = !inside;

	// 加载需要处理的ID范围
	uint2 range = ranges[block.group_index().y * horizontal_blocks + block.group_index().x];
	const int rounds = ((range.y - range.x + BLOCK_SIZE - 1) / BLOCK_SIZE);
	int toDo = range.y - range.x;

	// 分配共享内存用于批量数据获取
	__shared__ int collected_id[BLOCK_SIZE];
	__shared__ float2 collected_xy[BLOCK_SIZE];
	__shared__ float4 collected_conic_opacity[BLOCK_SIZE];
	__shared__ float collected_depth[BLOCK_SIZE];

	// 初始化辅助变量
	float T = 1.0f;
	uint32_t contributor = 0;
	uint32_t last_contributor = 0;
	float C[CHANNELS] = { 0 };
	float D = 0.0f;

	// 迭代处理所有批次
	for (int i = 0; i < rounds; i++, toDo -= BLOCK_SIZE)
	{
		// 如果整个块都完成了则退出
		int num_done = __syncthreads_count(done);
		if (num_done == BLOCK_SIZE)
			break;

		// 集体将高斯体数据从全局内存加载到共享内存
		int progress = i * BLOCK_SIZE + block.thread_rank();
		if (range.x + progress < range.y)
		{
			int coll_id = point_list[range.x + progress];
			collected_id[block.thread_rank()] = coll_id;
			collected_xy[block.thread_rank()] = points_xy_image[coll_id];
			collected_conic_opacity[block.thread_rank()] = conic_opacity[coll_id];
			collected_depth[block.thread_rank()] = depth[coll_id];
		}
		block.sync();

		// 处理当前批次中的每个高斯体
		for (int j = 0; !done && j < min(BLOCK_SIZE, toDo); j++)
		{
			contributor++;

			// 使用圆锥矩阵重采样
			float2 xy = collected_xy[j];
			float2 d = { xy.x - pixf.x, xy.y - pixf.y };
			float4 con_o = collected_conic_opacity[j];
			float power = -0.5f * (con_o.x * d.x * d.x + con_o.z * d.y * d.y) - con_o.y * d.x * d.y;
			if (power > 0.0f)
				continue;

			// 计算alpha值
			float alpha = min(0.99f, con_o.w * exp(power));
			if (alpha < 1.0f / 255.0f) {
				continue;
			}
			float test_T = T * (1 - alpha);
			if (test_T < 0.0001f)
			{
				done = true;
				continue;
			}

			// 累积颜色和深度值
			for (int ch = 0; ch < CHANNELS; ch++) {
				C[ch] += features[collected_id[j] * CHANNELS + ch] * alpha * T;
			}
			D += collected_depth[j] * alpha * T;
			// 记录高斯体被多少像素使用
			if (test_T > 0.5f) {
				atomicAdd(&(is_used[collected_id[j]]), 1);
			}

			T = test_T;
			last_contributor = contributor;
		}
	}

	// 将最终渲染结果写入输出缓冲区
	if (inside)
	{
		final_T[pix_id] = T;
		n_contrib[pix_id] = last_contributor;
		for (int ch = 0; ch < CHANNELS; ch++) {
			out_color[ch * H * W + pix_id] = C[ch] + T * bg_color[ch];
		}
		out_depth[pix_id] = D;
		out_opacity[pix_id] = 1 - T;
	}
}

// 渲染函数的CPU端包装器
void FORWARD::render(
	const dim3 grid, dim3 block,
	const uint2* ranges,
	const uint32_t* point_list,
	int W, int H,
	const float2* means2D,
	const float* colors,
	const float4* conic_opacity,
	float* final_T,
	uint32_t* n_contrib,
	const float* bg_color,
	float* out_color,
	const float* depth,
	float* out_depth, 
	float* out_opacity,
	int* is_used)
{
	renderCUDA<NUM_CHANNELS> << <grid, block >> > (
		ranges,
		point_list,
		W, H,
		means2D,
		colors,
		conic_opacity,
		final_T,
		n_contrib,
		bg_color,
		out_color,
		depth,
		out_depth,
		out_opacity,
		is_used);
}

// 预处理函数的CPU端包装器
void FORWARD::preprocess(int P, int D, int M,
	const float* means3D,
	const glm::vec3* scales,
	const float scale_modifier,
	const glm::vec4* rotations,
	const float* opacities,
	const float* shs,
	bool* clamped,
	const float* cov3D_precomp,
	const float* colors_precomp,
	const float* viewmatrix,
	const float* projmatrix,
	const glm::vec3* cam_pos,
	const int W, int H,
	const float focal_x, float focal_y,
	const float tan_fovx, float tan_fovy,
	int* radii,
	float2* means2D,
	float* depths,
	float* cov3Ds,
	float* rgb,
	float4* conic_opacity,
	const dim3 grid,
	uint32_t* tiles_touched,
	bool prefiltered)
{
	preprocessCUDA<NUM_CHANNELS> << <(P + 255) / 256, 256 >> > (
		P, D, M,
		means3D,
		scales,
		scale_modifier,
		rotations,
		opacities,
		shs,
		clamped,
		cov3D_precomp,
		colors_precomp,
		viewmatrix, 
		projmatrix,
		cam_pos,
		W, H,
		tan_fovx, tan_fovy,
		focal_x, focal_y,
		radii,
		means2D,
		depths,
		cov3Ds,
		rgb,
		conic_opacity,
		grid,
		tiles_touched,
		prefiltered
		);
}